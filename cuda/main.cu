#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#define t1 4096
#define t2 4096
#define N 1
#define ITERATIONS 10
#define BLOCK_SIZE 32
using namespace std;
float A[N * N], B[N * N], C[N * N], C_cmp[N * N];

__global__ void split(float *C11, float *C12, float *C21, float *C22, float *C, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C11[i * n + j] = C[i * 2 * n + j];
		C12[i * n + j] = C[i * 2 * n + j + n];
		C21[i * n + j] = C[(i + n) * 2 * n + j];
		C22[i * n + j] = C[(i + n) * 2 * n + j + n];
	}
}

__global__ void merge(float *C11, float *C12, float *C21, float *C22, float *C, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C[i * 2 * n + j] = C11[i * n + j];
		C[i * 2 * n + j + n] = C12[i * n + j];
		C[(i + n) *2 * n + j] = C21[i * n + j];
		C[(i + n) * 2 * n + j + n] = C22[i * n + j];
	}
}

__global__ void add(float *A, float *B, float *C, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C[i * n + j] = A[i * n + j] + B[i * n + j];
	}
}

__global__ void sub(float *A, float *B, float *C, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C[i * n + j] = A[i * n + j] - B[i * n + j];
	}
}

__global__ void mul(float *A, float *B, float *C, int n) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C[i * n + j] = 0;
		for(int k = 0; k < n; k++) {
			C[i * n + j] += A[i * n + k] * B[k * n + j];
		}
	}
}

__global__ void mul_add(float *A, float *B, float *T, float *C, int n) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C[i * n + j] = T[i * n + j];
		for(int k = 0; k < n; k++) {
			C[i * n + j] += A[i * n + k] * B[k * n + j];
		}
	}
}

__global__ void mul_sub_inc(float *A, float *B, float *T, float *C1, float *C2, int n) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C1[i * n + j] = 0;
		for(int k = 0; k < n; k++) {
			C1[i * n + j] += A[i * n + k] * B[k * n + j];
		}
		C1[i * n + j] = T[i * n + j] - C1[i * n + j];
		C2[i * n + j] += T[i * n + j];
	}
}

__global__ void mul_inc_inc_inc(float *A, float *B, float *C, float *T, float *C1, float *C2, int n) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	if(i < n && j < n) {
		C[i * n + j] = 0;
		for(int k = 0; k < n; k++) {
			C[i * n + j] += A[i * n + k] * B[k * n + j];
		}
		C1[i * n + j] += C[i * n + j];
		C2[i * n + j] += C1[i * n + j];
		C1[i * n + j] += T[i * n + j];
	}
}

void strassen(float *A, float *B, float *C, int n) {
	float *A_gpu, *B_gpu, *C_gpu;
	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	hipMalloc((void **)&A_gpu, sizeof(float) * n * n);
	hipMalloc((void **)&B_gpu, sizeof(float) * n * n);
	hipMalloc((void **)&C_gpu, sizeof(float) * n * n);
	hipMemcpy(A_gpu, A, sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(float) * n * n, hipMemcpyHostToDevice);
	if(n <= t1) {
		dim3 grid((size_t)ceil((float)n / (float)block.x), (size_t)ceil((float)n / (float)block.y));
		mul<<<grid, block>>>(A_gpu, B_gpu, C_gpu, n);
		hipDeviceSynchronize();
	}
	else{
		int m = n >> 1;
		dim3 grid((size_t)ceil((float)m / (float)block.x), (size_t)ceil((float)m / (float)block.y));
		float *A11, *A12, *A21, *A22, *B11, *B12, *B21, *B22, *C11, *C12, *C21, *C22, *T1, *T2;
		hipMalloc((void **)&A11, sizeof(float) * m * m);
		hipMalloc((void **)&A12, sizeof(float) * m * m);
		hipMalloc((void **)&A21, sizeof(float) * m * m);
		hipMalloc((void **)&A22, sizeof(float) * m * m);
		hipMalloc((void **)&B11, sizeof(float) * m * m);
		hipMalloc((void **)&B12, sizeof(float) * m * m);
		hipMalloc((void **)&B21, sizeof(float) * m * m);
		hipMalloc((void **)&B22, sizeof(float) * m * m);
		hipMalloc((void **)&C11, sizeof(float) * m * m);
		hipMalloc((void **)&C12, sizeof(float) * m * m);
		hipMalloc((void **)&C21, sizeof(float) * m * m);
		hipMalloc((void **)&C22, sizeof(float) * m * m);
		hipMalloc((void **)&T1, sizeof(float) * m * m);
		hipMalloc((void **)&T2, sizeof(float) * m * m);
		if(n <= t2) {
			split<<<grid, block>>>(A11, A12, A21, A22, A_gpu, m);
			hipDeviceSynchronize();
			split<<<grid, block>>>(B11, B12, B21, B22, B_gpu, m);
			hipDeviceSynchronize();
			sub<<<grid, block>>>(A11, A21, T1, m);
			hipDeviceSynchronize();
			sub<<<grid, block>>>(B22, B12, T2, m);
			hipDeviceSynchronize();
			mul<<<grid, block>>>(T1, T2, C21, m);
			hipDeviceSynchronize();
			add<<<grid, block>>>(A21, A22, T1, m);
			hipDeviceSynchronize();
			sub<<<grid, block>>>(B12, B11, T2, m);
			hipDeviceSynchronize();
			mul<<<grid, block>>>(T1, T2, C22, m);
			hipDeviceSynchronize();
			sub<<<grid, block>>>(T1, A11, T1, m);
			hipDeviceSynchronize();
			sub<<<grid, block>>>(B22, T2, T2, m);
			hipDeviceSynchronize();
			mul<<<grid, block>>>(T1, T2, C11, m);
			hipDeviceSynchronize();
			sub<<<grid, block>>>(A12, T1, T1, m);
			hipDeviceSynchronize();
			mul_add<<<grid, block>>>(T1, B22, C22, C12, m);
			hipDeviceSynchronize();
			mul_inc_inc_inc<<<grid, block>>>(A11, B11, T1, C21, C11, C12, m);
			hipDeviceSynchronize();
			sub<<<grid, block>>>(T2, B21, T2, m);
			hipDeviceSynchronize();
			mul_sub_inc<<<grid, block>>>(A22, T2, C11, C21, C22, m);
			hipDeviceSynchronize();
			mul_add<<<grid, block>>>(A12, B21, T1, C11, m);
			hipDeviceSynchronize();
			merge<<<grid, block>>>(C11, C12, C21, C22, C_gpu, m);	
			hipDeviceSynchronize();
		}
		else{
			split<<<grid, block>>>(A11, A12, A21, A22, A_gpu, m);
			hipDeviceSynchronize();
			split<<<grid, block>>>(B11, B12, B21, B22, B_gpu, m);
			hipDeviceSynchronize();
			sub<<<grid, block>>>(A11, A21, T1, m);
			hipDeviceSynchronize();
			sub<<<grid, block>>>(B22, B12, T2, m);
			hipDeviceSynchronize();
			strassen(T1, T2, C21, m);
			add<<<grid, block>>>(A21, A22, T1, m);
			hipDeviceSynchronize();
			sub<<<grid, block>>>(B12, B11, T2, m);
			hipDeviceSynchronize();
			strassen(T1, T2, C22, m);
			sub<<<grid, block>>>(T1, A11, T1, m);
			hipDeviceSynchronize();
			sub<<<grid, block>>>(B22, T2, T2, m);
			hipDeviceSynchronize();
			strassen(T1, T2, C11, m);
			sub<<<grid, block>>>(A12, T1, T1, m);
			hipDeviceSynchronize();
			strassen(T1, B22, C12, m);
			add<<<grid, block>>>(C12, C22, C12, m);
			hipDeviceSynchronize();
			strassen(A11, B11, T1, m);
			add<<<grid, block>>>(C11, C12, C12, m);
			hipDeviceSynchronize();
			add<<<grid, block>>>(C12, T1, C12, m);
			hipDeviceSynchronize();
			add<<<grid, block>>>(C11, C21, C11, m);
			hipDeviceSynchronize();
			add<<<grid, block>>>(C11, T1, C11, m);
			hipDeviceSynchronize();
			sub<<<grid, block>>>(T2, B21, T2, m);
			hipDeviceSynchronize();
			strassen(A22, T2, C21, m);
			sub<<<grid, block>>>(C11, C21, C21, m);
			hipDeviceSynchronize();
			add<<<grid, block>>>(C11, C22, C22, m);
			hipDeviceSynchronize();
			strassen(A12, B21, C11, m);
			add<<<grid, block>>>(C11, T1, C11, m);
			hipDeviceSynchronize();
			merge<<<grid, block>>>(C11, C12, C21, C22, C_gpu, m);	
			hipDeviceSynchronize();
		}
		hipFree(A11); 
		hipFree(A12); 
		hipFree(A21); 
		hipFree(A22); 
		hipFree(B11); 
		hipFree(B12); 
		hipFree(B21); 
		hipFree(B22); 
		hipFree(T1);
		hipFree(T2);	
	} 
	hipMemcpy(C, C_gpu, sizeof(float) * n * n, hipMemcpyDeviceToHost);
	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
}

void compare(float *res1, float *res2, int n) {
	int fail = 0;
	for(int i = 0; i < n; i++) {
		float a, b;
		a = res1[i] < 0 ? -res1[i] : res1[i];
		b = res2[i] < 0 ? -res2[i] : res2[i];
		if(a < 0.01 && b < 0.01) {
			continue;
		}
		if(i<10) {
			printf("i = %d\t%lf\t%lf\n", i, a, b);
		}
		float diff = (a - b) / (a + 0.000001);
		if(diff < 0) {
			diff= -diff;
		}
		if(diff>0.0005) {
			fail++;
		}
	}
	printf("Number of errors: %d\n", fail);
}

double timestamp(){
	struct timeval tv;
	gettimeofday(&tv, 0);
	return tv.tv_sec + 1e-6 * tv.tv_usec;
}

int main() {
	for(int i = 0; i < N; i++) {
		for(int j = 0; j < N; j++) {
			A[i * N + j] = (float)rand() / (float)RAND_MAX;
			B[i * N + j] = (float)rand() / (float)RAND_MAX;
			C[i * N + j] = 0;
			C_cmp[i * N + j] = 0;
		}
	}

	for(int j = 0; j < N; j++) {
		for(int i = 0; i < N; i++) {
			for(int k = 0; k < N; k++) {
				C_cmp[i * N + j] += A[i * N + k] * B[k * N + j];
			}
		}
	}
	strassen(A, B, C, N);
	compare(C, C_cmp, N * N);

	double time1 = timestamp();
	for(int numOfTimes = 0; numOfTimes < ITERATIONS; numOfTimes++) {
		strassen(A, B, C, N);
	}
	double time2 = timestamp();

	double time = (time2 - time1) / ITERATIONS;
	double flops = 2.0 * N * N * N;
	double gflopsPerSecond = flops / 1000000000 /time;
	double GB = 4.0 * N * N / 1000000000;
	double GBpS = 4.0 * N * N / 1000000000 / time;
	printf("GFLOPS/s = %lf\n", gflopsPerSecond);
	printf("GB/s = %lf\n", GBpS);
	printf("GFLOPS = %lf\n", flops / 1000000000);
	printf("GB = %lf\n", GB);
	printf("time(s) = %lf\n", time);
	return 0;
}
